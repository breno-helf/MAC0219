#include "hip/hip_runtime.h"
#include "io.hpp"
#include "min.hpp"
#include "error_handler.hpp"
#include <iostream>

using namespace std;

int main() {
    Matrices a = readMatricesFromFile("asd.txt");
    
//    for (int k = 0; k < a.length; k += 9) {
//        for (int i = 0; i < 9; i++) {
//            if (i % 3 == 0) cout << '\n';
//            cout << a.v[k + i] << " ";
//        }
//        cout << endl;
//    }
//    
//    sequentialReduction(a);
//
// 
//    cout << "RESULT: \n\n";
//    for (int i = 0; i < 9; i++) {
//        if (i % 3 == 0) cout << '\n';
//        cout << a.v[i] << " ";
//    }
//    cout << endl;

//    delete[] a.v;

  
    a = readMatricesFromFile("asd.txt");
    
    for (int i = 0; i < a.length; i++) {
        cout << a.v[i] << " ";
    }
    cout << endl;
    
    allocateMatricesToCuda(&a);

    cout << endl;

    
    cudaReduction<<<1, 288>>>(a);
    errorCheck();

    cout << "ALO --> " << a.dv[0] << '\n';
    cout << endl;

    return 0;
}
