#include "hip/hip_runtime.h"
#include "io.hpp"
#include "min.hpp"
#include <iostream>

using namespace std;

int main() {
    Matrices a = readMatricesFromFile("asd.txt");
    
    for (int k = 0; k < a.length; k += 9) {
        for (int i = 0; i < 9; i++) {
            if (i % 3 == 0) cout << '\n';
            cout << a.v[k + i] << " ";
        }
    }
    
    sequentialReduction(a);

    cout << "RESULT: \n\n";
    for (int i = 0; i < 9; i++) {
        if (i % 3 == 0) cout << '\n';
        cout << a.v[i] << " ";
    }

    delete[] a.v;
    
    a = readMatricesFromFile("asd.txt");
    allocateMatricesToCuda(a);

    for (int i = 0; i < a.length; i++) {
        cout << a.v[i] << ' ';
    }
    cout << endl;

    
    cudaReduction<<<1, 1>>>(a);

    cout << "ALO --> " << a.v[0] << '\n';
    cout << endl;

}
