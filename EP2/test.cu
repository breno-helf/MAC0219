#include "hip/hip_runtime.h"
#include "io.hpp"
#include "min.hpp"
#include "error_handler.hpp"
#include <iostream>
#include <ctime>
#include <unistd.h>
using namespace std;

int main(int argc, char ** argv) {
    if (argc < 2) {
        fprintf(stderr, "Not enough arguments");
        return 1;
    }

    printf("Reading Matrices\n");

    char * matrixFile = argv[1];
    Matrices a = readMatricesFromFile(matrixFile);
     
    allocateMatricesToCuda(a);

    printf("Matrixes in memory\n");

    printf("there are %d matrices\n", a.length);
    
    clock_t begin, end;
    bool same_ans = true;
    float elapsed = 0;

    begin = clock();
    int * cudaAns = cudaReduceMatrix(a);
    hipDeviceSynchronize();
    end = clock();

    elapsed = (end - begin) / CLOCKS_PER_SEC;
        
    printf("Cuda ans -- %.10f Seconds\n", elapsed);
    for (int i = 0; i < 9; i++) {
        printf("%d ", cudaAns[i]);
    }

    begin = clock();
    int * seqAns  = sequentialReductionMatrix(a);
    hipDeviceSynchronize();
    end = clock();

    elapsed = (end - begin) / CLOCKS_PER_SEC;
    
    printf("\n\nSequential ans -- %.10f Seconds\n", elapsed);
    for (int i = 0; i < 9; i++) {
        printf("%d ", seqAns[i]);
        if (seqAns[i] != cudaAns[i]) same_ans = false;
    }
    printf("\n\n");
    
    printf("%s\n", (same_ans ? "Cuda is CORRECT" : "Cuda is WRONG"));

    return 0;
}
