#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
using namespace std;
#include "min.hpp"
 
int magicMin(int a, int b) {
	return (a + b - abs(a - b)) >> 1;
}

void sequentialReduction(Matrices m) {
    for (int start = 0; start < 9; start++) {
         for (int i = start; i < m.length; i += 9) {
            m.v[start] = magicMin(m.v[start], m.v[i]);
         }
    }
}

__global__
void cudaReduction(Matrices m) {
    //unsigned int tid = threadIdx.x;

    for (int start = 0; start < 9; start++) {
        for (int i = start; i < m.length; i += 9) {
            m.dv[start] = min(m.dv[start], m.dv[i]);
            printf("JOVEM Min between %d %d --> %d\n", m.dv[start], m.dv[i], min(m.dv[start], m.dv[i]));
        }
    }

    printf("RESULT: \n\n");
    for (int i = 0; i < 9; i++) {
        if (i % 3 == 0) printf("\n");
        printf("%d ", m.dv[i]);
    }

    printf("\n");
}
