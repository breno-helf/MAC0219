#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
using namespace std;
#include "min.hpp"
 
int magicMin(int a, int b) {
	return (a + b - abs(a - b)) >> 1;
}

void sequentialReduction(Matrices m) {
    for (int start = 0; start < 9; start++) {
         for (int i = start; i < m.length; i += 9) {
            m.v[start] = magicMin(m.v[start], m.v[i]);
         }
    }
}

__global__
void cudaReduction(Matrices m) {
    // each thread loads one element from global to shared mem
    int tid = threadIdx.x;
    int n = m.length;
    int start = (tid * n) / 288;
    int end = (tid * (n + 1)) / 288;
    
    for (int s = start; s < end; s++) {
        m.dv[start] = min(m.dv[start], m.dv[end]);
    }    

    __syncthreads();

    if (tid == 0) {
        for (int k = 0; k < 288; k++) {
            m.dv[0] = min(m.dv[0], m.dv[(k * n) / 288] );
        }
    }

    __syncthreads();
}
