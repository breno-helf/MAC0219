#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
using namespace std;
#include "min.hpp"
 
int magicMin(int a, int b) {
	return (a + b - abs(a - b)) >> 1;
}

void sequentialReduction(Matrices m) {
    for (int start = 0; start < 9; start++) {
         for (int i = start; i < m.length; i += 9) {
             cout << "Min between " << m.v[start] << " " << m.v[i] << " --> ";
             cout << "Is " << magicMin(m.v[start], m.v[i]) << '\n';
             m.v[start] = magicMin(m.v[start], m.v[i]);
         }
    }
}

__global__
void cudaReduction(Matrices m) {
    //unsigned int tid = threadIdx.x;

    for (int start = 0; start < 9; start++) {
        for (int i = start; i < m.length; i += 9) {
            m.v[start] = min(m.v[start], m.v[i]);
            printf("JOVEM Min between %d %d --> %d\n", m.v[start], m.v[i], min(m.v[start], m.v[i]));
        }
    }

}
