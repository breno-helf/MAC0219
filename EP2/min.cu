#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
using namespace std;
#include "min.hpp"
 
int magicMin(int a, int b) {
	return (a + b - abs(a - b)) >> 1;
}

void sequentialReduction(Matrices m) {
    for (int start = 0; start < 9; start++) {
         for (int i = start; i < m.length; i += 9) {
            m.v[start] = magicMin(m.v[start], m.v[i]);
         }
    }
}

__global__
void cudaReduction(Matrices m) {
    // each thread loads one element from global to shared mem
    __shared__ int sdata[2 * 288];
    
    int tid = threadIdx.x;
    int globalId = blockIdx.x*blockDim.x + threadIdx.x;
    int n = m.length;
    int start = 2 * blockIdx.x * blockDim.x;

    if ((start + tid) < n) {
        sdata[tid] = m.dv[start + tid];
    }
    else {
        sdata[tid] = 0.0;
    }
    
    if ((start + blockDim.x + tid) < n) {
        sdata[blockDim.x + tid] = m.dv[start + blockDim.x + tid];
    }
    else {
        sdata[blockDim.x + tid] = 0.0;
    }

    // Traverse reduction tree
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
        __syncthreads();
        if (tid < stride)
            sdata[tid] = min(sdata[tid], sdata[tid + stride]);
    }
    __syncthreads();
    
    // Write the computed sum of the block to the output vector at correct index
    if (tid == 0 && (globalId*2) < n) {
        m.dv[blockIdx.x] = sdata[tid];
    }
}
