#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
using namespace std;
#include "min.hpp"
 
int magicMin(int a, int b) {
	return (a + b - abs(a - b)) >> 1;
}

void sequentialReduction(Matrices m) {
    for (int start = 0; start < 9; start++) {
         for (int i = start; i < m.length; i += 9) {
            m.v[start] = magicMin(m.v[start], m.v[i]);
         }
    }
}

__global__
void cudaReduction(Matrices m) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m.length)
        sdata[tid] = m.dv[i];
    __syncthreads();
    
    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // write result for this block to global mem\
    if (blockIdx.x < m.length)
        if (tid == 0) m.dv[blockIdx.x] = sdata[0];
    
}
