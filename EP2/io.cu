#include <sstream>
#include <fstream>
#include <string>
#include <stdexcept>
#include "io.hpp"

#define debug asm("int $3")

using namespace std;

Matrices readMatricesFromFile(const char *fileName) {
    string line;
    ifstream infile(fileName);
    // The state of the parser.
    // 0: waiting for number of matrices
    // 1: waiting for matrices numbers
    // 2: waiting for asterisks
    // 3: waiting for EOF
    int state = 0;
    int matrixLine = 0;
    long long fileLine = 0;
    // 2d matrices flattened in a vector
    int *v;
    long long index = 0;
    long long numberOfMatrices, currentMatrix = -1;
    bool doContinue = true;
    stringstream err;
    while (getline(infile, line) && doContinue) {
        fileLine++;
        istringstream it(line);
        switch (state) {
            case 0: {
                long long n;
                if (!(it >> n)) {
                    err << "readMatricesFromFile:" << fileLine
                        << ": Expected number";
                    throw runtime_error(err.str());
                }
                else {
                    numberOfMatrices = n;
                    // 9 numbers per matrix
                    v = new int[9 * n];
                    state = 2;
                }
                break;
            }
            case 1: {
                int a, b, c;
                if (!(it >> a >> b >> c)) {
                    err << "readMatricesFromFile:" << fileLine
                    << ": Expected number";
                    throw runtime_error(err.str());
                }
                else {
                    v[index] = a;
                    v[index + 1] = b;
                    v[index + 2] = c;
                    index += 3;
                    if (matrixLine == 2) {
                        // go to asterisks
                        matrixLine = 0;
                        state = 2;
                    }
                    else {
                        matrixLine++;
                    }
                }
                break;
            }
            case 2: {
                // asterisks
                string ask;
                if (!(it >> ask) || ask.compare("***") != 0) {
                    err << "readMatricesFromFile:" << fileLine
                        << ": Expected '***', got " << ask;
                    throw runtime_error(err.str());
                }
                if (currentMatrix >= numberOfMatrices - 1) {
                    state = 3;
                }
                else {
                    currentMatrix++;
                    state = 1;
                }
                break;
            }
            case 3: {
                // Throws the rest away.
                doContinue = false;
                break;
            }
            default: {
                err << "readMatricesFromFile:" << fileLine
                    << ": invalid parser state: " << state;
                throw runtime_error(err.str());
            }
        }
    }
    if (state != 3) {
        err << "readMatricesFromFile:" << fileLine
            << ": could not read matrices from " << fileName;
        throw runtime_error(err.str());
    }
    Matrices result;
    result.v = v;
    result.length = 9 * numberOfMatrices;
    return result;
}

void allocateMatricesToCuda(Matrices m) {
    int *otherV;
    hipMallocManaged(&otherV, m.length * sizeof(int));
    for (long long i = 0; i < m.length; i++) {
        otherV[i] = m.v[i];
    }
    delete[] m.v;
    m.v = otherV;
}
